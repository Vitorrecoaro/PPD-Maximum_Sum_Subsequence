#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

// # qtdTotalSum = A quantidade total de somas parciais que serão feitas, que pode ser calculada por uma P.A de ordem 1
// # iniciando de "col" e indo até "n-1", sendo assim ficando a seguinte equação para o cálculo da soma total:
// # ((col + n - 1)*(n - col)) / 2 =>  n^2 + col - (n + col^2) / 2 
__global__ void calculatePartialSum( int totalOperations, int lin, int n, int *arr, int *dp){
	// # Para achar a qual elemento do vetor a thread se refere, é fazer a conta reversa, pois se o id da thread for 5,
	// # então já se passaram 5 contas feitas antes dela, assim podemos usar bhaskara para achar a qual elemento da arr 
	// # a thread se refere com a seguinte conta: "valorVet"=(1 + sqrt(1-4*(lin - lin^2 - 2*idx))/2, como poderá dar uns
    // # quebrados, pegamos o chão desta conta.

	// # E para achar com qual elemento da array ela vai tentar fazer a soma a gente usa o resultado anterior e calcula 
	// # em qual thread id se iniciou aquele bloco de comparação e faz a subtração desse elemento com o thread id. 
	// # Utilizando a seguinte conta: threadId - (valorVet^2 + lin - valorVet - lin^2)/ 2.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < totalOperations){
        int refValue = (int)(1 + sqrtf((float)(1 - 4 * ((lin + 1) - ((lin + 1) * (lin + 1)) - 2 * idx)))) / 2;
        int compValue = (int)(idx) - (int)(refValue * refValue + (lin + 1) - refValue - (lin + 1) * (lin + 1)) / 2;

        if(arr[compValue] < arr[refValue]){
            if(dp[lin * n + compValue] != -1){
                int newVal = dp[lin * n + compValue] + arr[refValue];
                atomicMax(&dp[(lin + 1) * n + refValue ], newVal);
            }
        }
    }
}

__global__ void calculateAnswer(int n, int k, int *ans, int *dp){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n){
        if (*ans < dp[ (k - 1) * n + idx]){
            atomicMax(ans, dp[ (k - 1) * n + idx]);
        }
    }
}

int MaxIncreasingSub(int *arr, int n, int k) 
{
	int *dp = (int *)malloc(n * k * sizeof(int));
    for (int i = 0; i < n * k; i++) {
        dp[i] = -1;
    }

    for (int i = 0; i < n; i++) {
        dp[i] = arr[i];
    }

    int *d_arr, *d_dp;

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_dp, n * k * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dp, dp, n * k * sizeof(int), hipMemcpyHostToDevice);
    free(dp);

	for(int i = 0; i < (k - 1); i ++){
        int threadsPerBlock = 1024;
        int totalOperations = (n * n + (i + 1) - (n + (i + 1) * (i + 1))) / 2;
        int blocksPerGrid = (totalOperations + threadsPerBlock - 1) / threadsPerBlock;
        calculatePartialSum<<<blocksPerGrid, threadsPerBlock>>>( totalOperations, i, n, d_arr, d_dp);
        hipDeviceSynchronize();
	}
    hipFree(d_arr);

    int ans, *d_ans;

    ans = -1;
    hipMalloc((void **)&d_ans, sizeof(int));
    hipMemcpy(d_ans, &ans, sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = ( n + threadPerBlock - 1 ) / threadPerBlock;
    calculateAnswer<<<blockPerGrid, threadPerBlock>>>(n, k, d_ans, d_dp);
    hipDeviceSynchronize();
    hipMemcpy(&ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_dp);
    hipFree(d_ans);

    return (ans == -1) ? 0 : ans; 
} 

int main() 
{ 
	int n, k;
    scanf(" %d", &n);
    scanf(" %d", &k);
    int *arr = (int *) malloc(n * sizeof(int));
	for (int i = 0; i<n; i++)
		scanf(" %d", &arr[i]);
	int ans = MaxIncreasingSub(arr, n, k); 
	printf("%d\n", ans); 
	return 0;
} 
