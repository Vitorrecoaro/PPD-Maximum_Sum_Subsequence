#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

__global__ void calculatePartialSum( int totalOperations, int lin, int n, int *arr, int *dp){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < totalOperations){
        int refValue = (int)(1 + sqrtf((float)(1 - 4 * ((lin + 1) - ((lin + 1) * (lin + 1)) - 2 * idx)))) / 2;
        int compValue = (int)(idx) - (int)(refValue * refValue + (lin + 1) - refValue - (lin + 1) * (lin + 1)) / 2;

        if(arr[compValue] < arr[refValue]){
            if(dp[lin * n + compValue] != -1){
                int newVal = dp[lin * n + compValue] + arr[refValue];
                atomicMax(&dp[(lin + 1) * n + refValue ], newVal);
            }
        }
    }
}

__global__ void calculateAnswer(int n, int k, int *ans, int *dp){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n){
        if (*ans < dp[ (k - 1) * n + idx]){
            atomicMax(ans, dp[ (k - 1) * n + idx]);
        }
    }
}

int MaxIncreasingSub(int *arr, int n, int k) 
{
	int *dp = (int *)malloc(n * k * sizeof(int));
    for (int i = 0; i < n * k; i++) {
        dp[i] = -1;
    }

    for (int i = 0; i < n; i++) {
        dp[i] = arr[i];
    }

    int *d_arr, *d_dp;

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_dp, n * k * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dp, dp, n * k * sizeof(int), hipMemcpyHostToDevice);
    free(dp);

	for(int i = 0; i < (k - 1); i ++){
        int threadsPerBlock = 1024;
        int totalOperations = (n * n + (i + 1) - (n + (i + 1) * (i + 1))) / 2;
        int blocksPerGrid = (totalOperations + threadsPerBlock - 1) / threadsPerBlock;
        calculatePartialSum<<<blocksPerGrid, threadsPerBlock>>>( totalOperations, i, n, d_arr, d_dp);
        hipDeviceSynchronize();
	}
    hipFree(d_arr);

    int ans, *d_ans;

    ans = -1;
    hipMalloc((void **)&d_ans, sizeof(int));
    hipMemcpy(d_ans, &ans, sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = ( n + threadPerBlock - 1 ) / threadPerBlock;
    calculateAnswer<<<blockPerGrid, threadPerBlock>>>(n, k, d_ans, d_dp);
    hipDeviceSynchronize();
    hipMemcpy(&ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_dp);
    hipFree(d_ans);

    return (ans == -1) ? 0 : ans; 
} 

int main() 
{ 
	int n, k;
    scanf(" %d", &n);
    scanf(" %d", &k);
    int *arr = (int *) malloc(n * sizeof(int));
	for (int i = 0; i<n; i++)
		scanf(" %d", &arr[i]);
	int ans = MaxIncreasingSub(arr, n, k); 
	printf("%d\n", ans); 
	return 0;
} 
