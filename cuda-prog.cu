#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

// # qtdTotalSum = A quantidade total de somas parciais que serão feitas, que pode ser calculada por uma P.A de ordem 1
// # iniciando de "col" e indo até "n-1", sendo assim ficando a seguinte equação para o cálculo da soma total:
// # ((col + n - 1)*(n - col)) / 2 =>  n^2 + col - (n + col^2) / 2 
__global__ void calculatePartialSum( int totalOperations, int col, int k, int *arr, int *dp){
	// # Para achar a qual elemento do vetor a thread se refere, é fazer a conta reversa, pois se o id da thread for 5,
	// # então já se passaram 5 contas feitas antes dela, assim podemos usar bhaskara para achar a qual elemento da arr 
	// # a thread se refere com a seguinte conta: "valorVet"=(1 + sqrt(1-4*(col - col^2 - 2*idx))/2, como poderá dar uns
    // # quebrados, pegamos o chão desta conta.

	// # E para achar com qual elemento da array ela vai tentar fazer a soma a gente usa o resultado anterior e calcula 
	// # em qual thread id se iniciou aquele bloco de comparação e faz a subtração desse elemento com o thread id. 
	// # Utilizando a seguinte conta: threadId - (valorVet^2 + col - valorVet - col^2)/ 2.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < totalOperations){
        int refValue = (int)(1 + sqrtf((float)(1 - 4 * (col - (col * col) - 2 * idx)))) / 2;
        int compValue = (int)(idx) - (int)(refValue * refValue + col - refValue - col * col) / 2;

        if(arr[compValue] < arr[refValue]){
            if(dp[compValue * (k + 1) + col] != -1){
                int newVal = dp[compValue * (k + 1) + col] + arr[refValue];
                atomicMax(&dp[refValue * (k + 1) + col + 1], newVal);
            }
        }
    }
}

__global__ void calculateAnswer(int n, int k, int *ans, int *dp){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n){
        if (*ans < dp[ idx * (k + 1) + k]){
            atomicMax(ans, dp[ idx * (k + 1) + k]);
        }
    }
}

int MaxIncreasingSub(int *arr, int n, int k) 
{
	int *dp = (int *)malloc(n * (k + 1) * sizeof(int));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k + 1; j++) {
            dp[i * (k + 1) + j] = -1;
        }
    }

    for (int i = 0; i < n; i++) {
        dp[i * (k + 1) + 1] = arr[i];
    }

    int *d_arr, *d_dp;

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_dp, n * (k + 1) * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dp, dp, n * (k + 1) * sizeof(int), hipMemcpyHostToDevice);
    free(dp);

	for(int i = 1; i < k; i ++){
        int threadsPerBlock = 1024;
        int totalOperations = (n * n + i - (n + i * i)) / 2;
        int blocksPerGrid = (totalOperations + threadsPerBlock - 1) / threadsPerBlock;
        calculatePartialSum<<<blocksPerGrid, threadsPerBlock>>>( totalOperations, i, k, d_arr, d_dp);
        hipDeviceSynchronize();
	}
    hipFree(d_arr);

    int ans, *d_ans;

    ans = -1;
    hipMalloc((void **)&d_ans, sizeof(int));
    hipMemcpy(d_ans, &ans, sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = ( n + threadPerBlock - 1 ) / threadPerBlock;
    calculateAnswer<<<blockPerGrid, threadPerBlock>>>(n, k, d_ans, d_dp);
    hipDeviceSynchronize();
    hipMemcpy(&ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_dp);
    hipFree(d_ans);

    return (ans == -1) ? 0 : ans; 
} 

int main() 
{ 
	int n, k;
    scanf(" %d", &n);
    scanf(" %d", &k);
    int *arr = (int *) malloc(n * sizeof(int));
	for (int i = 0; i<n; i++)
		scanf(" %d", &arr[i]);
	int ans = MaxIncreasingSub(arr, n, k); 
	printf("%d\n", ans); 
	return 0;
} 
