#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

// # qtdTotalSum = A quantidade total de somas parciais que serão feitas, que pode ser calculada por uma P.A de ordem 1
// # iniciando de "col" e indo até "n-1", sendo assim ficando a seguinte equação para o cálculo da soma total:
// # ((col + n - 1)*(n - col)) / 2 =>  n^2 + col - (n + col^2) / 2 
__global__ void calculatePartialSum( int totalOperations, int col, int k, int *arr, int *dp){
	// # Para achar a qual elemento do vetor a thread se refere, é fazer a conta reversa, pois se o id da thread for 5,
	// # então já se passaram 5 contas feitas antes dela, assim podemos usar bhaskara para achar a qual elemento da arr 
	// # a thread se refere com a seguinte conta: "valorVet"=(1 + sqrt(1-4*(col - col^2 - 2*idx))/2, como poderá dar uns
    // # quebrados, pegamos o chão desta conta.

	// # E para achar com qual elemento da array ela vai tentar fazer a soma a gente usa o resultado anterior e calcula 
	// # em qual thread id se iniciou aquele bloco de comparação e faz a subtração desse elemento com o thread id. 
	// # Utilizando a seguinte conta: threadId - (valorVet^2 + col - valorVet - col^2)/ 2.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < totalOperations){
        int refValue = (int)(1 + sqrtf((float)(1-4*(col - (col * col) - 2 * idx))))/2;
        int compValue = (int)(idx) - (int)(refValue * refValue + col - refValue - col * col)/2;

        // printf("idx: %d, ref: %d, cmp: %d\n", idx, refValue, compValue);

        if(arr[compValue] < arr[refValue]){
            if(dp[compValue * (k + 1) + col] != -1){
                int newVal = dp[compValue * (k + 1) + col] + arr[refValue];
                atomicMax(&dp[refValue * (k + 1) + col + 1], newVal);
            }
        }
    }
}

int MaxIncreasingSub(int *arr, int n, int k) 
{
	int *dp = (int*)malloc(n * (k+1) * sizeof(int));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k+1; j++) {
            dp[i * (k+1) + j] = -1;
        }
    }

    for (int i = 0; i < n; i++) {
        dp[i * (k+1) + 1] = arr[i];
    }

    int *d_arr, *d_dp;
    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_dp, n * (k+1) * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dp, dp, n * (k+1) * sizeof(int), hipMemcpyHostToDevice);

	for(int i = 0; i < k; i ++){
        int threadsPerBlock = 1024;
        int totalOperations = (n*n + i - (n + i*i))/2;
        int blocksPerGrid = (totalOperations + threadsPerBlock - 1) / threadsPerBlock;
        calculatePartialSum<<<blocksPerGrid, threadsPerBlock>>>( totalOperations, i, k, d_arr, d_dp);
        hipDeviceSynchronize();
	}

    hipMemcpy(dp, d_dp, n * (k+1) * sizeof(int), hipMemcpyDeviceToHost);

    // for(int i = 0; i < n; i++){
    //     for(int j = 0; j < k + 1; j++){
    //         printf("%d ", dp[i * (k + 1) + j]);
    //     }
    //     printf("\n");
    // }
    
    hipFree(d_arr);
    hipFree(d_dp);

    int ans = -1;
    for (int i = 0; i < n; i++) {
        if (ans < dp[i * (k+1) + k]) {
            ans = dp[i * (k+1) + k];
        }
    }

    free(dp);
    return (ans == -1) ? 0 : ans; 
} 

int main() 
{ 
	int n, k;
    scanf(" %d", &n);
    scanf(" %d", &k);
    int *arr = (int *) malloc(n * sizeof(int));
	for (int i = 0; i<n; i++)
		scanf(" %d", &arr[i]);
	int ans = MaxIncreasingSub(arr, n, k); 
	printf("%d\n", ans); 
	return 0;
} 
